#include "hip/hip_runtime.h"
#include "../include/vertex_sequence.cuh"
#include <cassert>

VertexSequence::VertexSequence(int nv){
	p = new Point[nv];
	numVertices = nv;
}

__host__ __device__ VertexSequence::~VertexSequence(){
    if(p != nullptr){
        delete []p;
    }
}

box *VertexSequence::getMBR(){
	box *mbr = new box();
	for(int i = 0; i < numVertices; i ++){
		double lowx = min(mbr->get_lowx(), p[i].get_x());
		double highx = max(mbr->get_highx(), p[i].get_x());
		double lowy = min(mbr->get_lowy(), p[i].get_y());
		double highy = max(mbr->get_highy(), p[i].get_y());
		mbr->set_box(lowx, lowy, highx, highy);
	}
	return mbr;
}

int VertexSequence::get_numVertices(){return numVertices;}
double VertexSequence::get_pointX(int idx){return p[idx].get_x();}
double VertexSequence::get_pointY(int idx){return p[idx].get_y();}

size_t VertexSequence::decode(char *source){
	size_t decoded = 0;
	numVertices = ((size_t *)source)[0];
	assert(numVertices>0);
	p = new Point[numVertices + 1];
	decoded += sizeof(size_t);
	memcpy((char *)p,source+decoded,numVertices*sizeof(Point));
	decoded += numVertices*sizeof(Point);
	p[numVertices ++] = p[0];
	return decoded;
}