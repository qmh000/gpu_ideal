#include "hip/hip_runtime.h"
#include "../include/hip/hip_runtime_api.h"
#include "../include/MyPolygon.cuh"

int main(int argc, char** argv){
    query_context global_ctx;
    global_ctx.num_threads = 1;
    global_ctx.source_polygons = load_binary_file("/home/qmh/data/has_child.idl", global_ctx);
    
    //析构函数有问题
    preprocess(&global_ctx);

    printf("Rasterization Finished!\n");



    int size1 = global_ctx.source_polygons.size(), size2 = 0;
    MyRaster* h_rasters = new MyRaster[size1];
    Point* h_points = load_points("/home/qmh/data/sampled.points.dat", size2);

    int size = min(size1, size2);
    MyRaster* d_rasters = nullptr;
    Point* d_points = nullptr;
    int memsize1 = sizeof(MyRaster) * size;
    int memsize2 = sizeof(Point) * size;

    checkCudaErrors(hipMalloc((void**) &d_rasters, memsize1));
    checkCudaErrors(hipMalloc((void**) &d_points, memsize2));

    checkCudaErrors(hipMemcpy(d_points, h_points, memsize2, hipMemcpyHostToDevice));

    for(int i = 0; i < size; i ++){
        checkCudaErrors(hipMalloc((void **) &h_rasters[i].mbr, sizeof(box)));
        checkCudaErrors(hipMemcpy(h_rasters[i].mbr, global_ctx.source_polygons[i]->raster->mbr, sizeof(box), hipMemcpyHostToDevice));

        checkCudaErrors(hipMalloc((void **) &h_rasters[i].vs, sizeof(VertexSequence)));
        VertexSequence* h_vs = new VertexSequence();
        int vsNum = global_ctx.source_polygons[i]->get_numVertices();
        h_vs->numVertices = vsNum;
        checkCudaErrors(hipMalloc((void **) &h_vs->p, vsNum * sizeof(Point)));
        checkCudaErrors(hipMemcpy(h_vs, global_ctx.source_polygons[i]->raster->vs->p, vsNum * sizeof(Point), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(h_rasters[i].vs, h_vs, sizeof(VertexSequence), hipMemcpyHostToDevice));

        checkCudaErrors(hipMalloc((void **) &h_rasters[i].pixels, sizeof(Pixel)));
        
    }

    




}